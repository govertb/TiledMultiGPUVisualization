#include "hip/hip_runtime.h"
/*
 ==============================================================================

 mandelbrot.cu
 Author: Govert Brinkmann, unless a 'due' is given.

 This code was developed as part of research at the Leiden Institute of
 Advanced Computer Science (https://liacs.leidenuniv.nl).

 ==============================================================================
*/

// C
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <pthread.h>
#include <cpuid.h>

// C++
#include <iostream>
#include <fstream>
#include <vector>
#include <thread>
#include <string>
#include <chrono>
#include <map>

// Window System
#include <X11/Xlib.h>
#include <X11/Xlib-xcb.h>
#include <xcb/xcb.h>
#include <xcb/glx.h>

// Graphics Library
#include "../lib/glad/glad.h"
#include "../lib/glad/glad_glx.h"

// Utility functions
#include "x11_util.h"
#include "gl_util.h"

// CUDA
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime_api.h>

#include "cuda_util.h"

// Mandelbrot & Benchmarking
#include "benchmark_setup.hpp"

// application state
bool program_should_run = true;
Display *dpy; // Xlib handle to X
xcb_connection_t *xcb_connection; // xcb handle to X
std::vector<pthread_t> threads;
pthread_barrier_t swap_barrier;

// what to render
typedef struct view_parameters
{} view_parameters_t;
view_parameters_t vpar_render, vpar_event;

// display/X info
int display_w = 0;
int display_h = 0;
std::map<int, xcb_screen_t> x_screens;
std::map<int, int> gpu_screen; // gpu -> screen_no map
const std::vector<int> screen_order = {0, 2, 1}; // for BigEye

// for each screen (screen_no as key):
std::map<int, int> swap_ready;
typedef struct screen_resources
{
    xcb_window_t x_window;
    GLXWindow glx_window;
    GLXContext gl_context;
    GLXFBConfig* fb_configs;
    int gpu_id;
    hipGraphicsResource_t cuda_pbo_resource;
} screen_resources_t;
std::map<int, screen_resources_t> display_screen_resources;
std::map<xcb_window_t, int> window_to_screen_no;

// per screen, benchmarks
std::map<int, benchmark_clock_t::duration> inter_frame_durations;
std::map<int, std::vector<benchmark_clock_t::duration>> inter_frame_durations_benchmark;

// gpu properties and memory pointers, indexed by GPU ID.
std::vector<int> compute_gpu_ids;
std::map<int, dim3> bdims, gdims;

// function definitions
void init_xlib()
{
    if(!XInitThreads())
    {
        printf("Error: Xlib or machine does not support multithreading,\n"
               "       or initialization of multithreading failed.\n");
        exit(EXIT_FAILURE);
    }

    dpy = XOpenDisplay(NULL);
    if(dpy == NULL)
    {
        printf("Error: failed to connect to display '%s'.\n", getenv("DISPLAY"));
        exit(EXIT_FAILURE);
    }
}

void init_xcb()
{
    xcb_connection = XGetXCBConnection(dpy);
    if(!xcb_connection)
    {
        printf("Error: failed to open XCB connection.\n");
        exit(EXIT_FAILURE);
    }

    XSetEventQueueOwner(dpy, XCBOwnsEventQueue);
}

void load_glx(Display *dpy)
{
    int screen_no = 0;
    if(!gladLoadGLX(dpy, screen_no))
    {
        printf("Error: couldn't load GLX through glad.\n");
        exit(EXIT_FAILURE);
    }
}

bool gl_loaded = false;
void load_gl()
{
    if(!gladLoadGLLoader((GLADloadproc) glXGetProcAddress))
    {
        printf("Error: couldn't load GL through glad\n");
        exit(EXIT_FAILURE);
    }
    gl_loaded = true;
}

void init_screens()
{
    // Record information about screens into screens std::map
    xcb_screen_iterator_t screen_iterator = xcb_setup_roots_iterator(xcb_get_setup(xcb_connection));
    const int screen_count = screen_iterator.rem;
    if(screen_count < 1)
    {
        printf("Error: number of X screens <= 0.\n");
        exit(EXIT_FAILURE);
    }

    // Else, we have screens
    for(int screen_no = 0; screen_iterator.rem; xcb_screen_next(&screen_iterator), screen_no++)
    {
        xcb_screen_t screen = *screen_iterator.data;
        x_screens[screen_no] = screen;

        display_w += screen.width_in_pixels;
        display_h = screen.height_in_pixels;

        // Important to do this here, so size of map doesn't change when in use.
        swap_ready[screen_no] = 0;
        std::chrono::milliseconds d{1};
        inter_frame_durations[screen_no] = benchmark_clock_t::duration(d);
        inter_frame_durations_benchmark[screen_no].push_back(benchmark_clock_t::duration(d));
    }

    // Framebuffer attributes required to be supported
    const int fb_attrib_list[] = {
                            GLX_DOUBLEBUFFER, True,
                            GLX_RENDER_TYPE, GLX_RGBA_BIT,
                            GLX_DRAWABLE_TYPE, GLX_WINDOW_BIT | GLX_PBUFFER_BIT,
                            None};

    // Get best matching FBConfigs for each screen.
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        screen_resources_t &sr = display_screen_resources[screen_no];

        int num_matches = 0;
        sr.fb_configs = glXChooseFBConfig(dpy, screen_no, fb_attrib_list, &num_matches);
        if(num_matches <= 0 or not sr.fb_configs)
        {
            printf("Error couldn't find any framebuffer configurations for screen %d.\n", screen_no);
            exit(EXIT_FAILURE);
        }
    }
}

void create_xwindows()
{
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        const xcb_screen_t screen = it.second;
        screen_resources_t &sr = display_screen_resources[screen_no];

        const xcb_window_t parent = screen.root;
        const int pos_x = 0;
        const int pos_y = 0;
        const uint16_t border_width = 0;

        int visual_id;
        glXGetFBConfigAttrib(dpy, sr.fb_configs[0], GLX_VISUAL_ID , &visual_id);

        const xcb_colormap_t colormap = xcb_generate_id(xcb_connection);
        xcb_create_colormap(xcb_connection, XCB_COLORMAP_ALLOC_NONE, colormap, parent, visual_id);

        const uint32_t eventmask = XCB_EVENT_MASK_EXPOSURE |
                                   XCB_EVENT_MASK_KEY_PRESS|
                                   XCB_EVENT_MASK_BUTTON_PRESS |
                                   XCB_EVENT_MASK_BUTTON_MOTION |
                                   XCB_EVENT_MASK_POINTER_MOTION |
                                   XCB_EVENT_MASK_STRUCTURE_NOTIFY;
        const uint32_t valuelist[] = { eventmask, colormap, 0 };
        const uint32_t valuemask = XCB_CW_EVENT_MASK | XCB_CW_COLORMAP;

        sr.x_window = xcb_generate_id(xcb_connection);
        xcb_create_window(xcb_connection, XCB_COPY_FROM_PARENT, sr.x_window, parent,
            pos_x, pos_y, screen.width_in_pixels, screen.height_in_pixels,
            border_width, XCB_WINDOW_CLASS_INPUT_OUTPUT, visual_id, valuemask,
            valuelist);

        if(not sr.x_window)
        {
            printf("Error: couldn't create X window for screen %d.\n", screen_no);
            exit(EXIT_FAILURE);
        }

        window_to_screen_no[screen.root] = screen_no;
    }
}

void create_glxwindows()
{
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        screen_resources_t &sr = display_screen_resources[screen_no];

        // Obtain parent, the X Window.
        xcb_window_t parent_window = sr.x_window;
        sr.glx_window = glXCreateWindow(dpy, sr.fb_configs[0], parent_window, NULL);
        if(!sr.glx_window)
        {
            printf("Error: couldn't create GLXWindow for screen %d.\n", screen_no);
            exit(EXIT_FAILURE);
        }
    }
}

void init_gl()
{
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        screen_resources_t &sr = display_screen_resources[screen_no];

        // Create an OpenGL context.
        const GLXContext share_list = NULL;
        const Bool direct = True;
        int context_attributes[] =
        {
            GLX_CONTEXT_MAJOR_VERSION_ARB, 4,
            GLX_CONTEXT_MINOR_VERSION_ARB, 1,
            GLX_CONTEXT_PROFILE_MASK_ARB, GLX_CONTEXT_CORE_PROFILE_BIT_ARB,
            None
        };

        GLXContext gl_context = glXCreateContextAttribsARB(dpy, sr.fb_configs[0],
                                                           share_list, direct, context_attributes);

        if(gl_context != NULL && glXIsDirect(dpy, gl_context))
        {
            sr.gl_context = gl_context;
        }
        else
        {
            printf("Failed to create GL Context.\n");
            if (gl_context != NULL && !glXIsDirect(dpy, gl_context))
                printf("  no direct GL Context could be made.\n");
            exit(EXIT_FAILURE);
        }

        // Make context current and bind glx_window.
        const GLXDrawable drawable = sr.glx_window;
        Bool context_made_current = glXMakeCurrent(dpy, drawable, gl_context);
        if(!context_made_current)
        {
            printf("Couldn't make context current for screen %d\n", screen_no);
            exit(EXIT_FAILURE);
        }

        // Disable VBLANK sync.
        glXSwapIntervalEXT(dpy, drawable, 0);

        if(not gl_loaded) load_gl();

        // Get associated CUDA device
        int cuda_devicecount;
        cuda_check_error(hipGetDeviceCount(&cuda_devicecount));
        uint num_gl_associated_devices;
        int *gl_associated_devices = new int[cuda_devicecount];
        cuda_check_error(hipGLGetDevices(&num_gl_associated_devices,
                                          gl_associated_devices, cuda_devicecount,
                                          hipGLDeviceListAll));
        if (num_gl_associated_devices == 1)
        {
            sr.gpu_id = gl_associated_devices[0];
            gpu_screen[sr.gpu_id] = screen_no;
        }

        else if (num_gl_associated_devices > 1)
        {
            printf("Error: more then one GPU associated with GL context for screen %d\n", screen_no);
            exit(EXIT_FAILURE);
        }

        else
        {
            printf("Error: couldn't find GL device used for GL context of screen %d... ", screen_no);
            exit(EXIT_FAILURE);
        }

        /*

            // Initialize OpenGL here.

        */

        release_gl_context(dpy);
    }
}

void init_interop()
{
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        screen_resources_t &sr = display_screen_resources[screen_no];

        // Make context for `screen_no' current.
        if(!glXMakeCurrent(dpy, sr.glx_window, sr.gl_context))
        {
            printf("Error: couldn't make context current for screen %d.\n", screen_no);
            exit(EXIT_FAILURE);
        }

        // Select corresponding CUDA device
        cuda_check_error(hipSetDevice(sr.gpu_id));

        /*

            // Register OpenGL objects for this screen with CUDA runtime
            cuda_check_error(hipGraphicsGLRegisterBuffer(..., ..., ...));

        */

        // Done
        release_gl_context(dpy);
    }
}

typedef struct crlparameters_
{
    int screen_no;
    bool use_interop;
    int max_iterations;
    bool record_fps;
} crlparameters_t;

void *compute_render_loop(void* args)
{
    crlparameters_t p = *((crlparameters_t *) args);

    // Obtain information of screen to render to
    xcb_screen_t screen = x_screens.at(p.screen_no);
    screen_resources_t &sr = display_screen_resources.at(p.screen_no);
    int screen_count = xcb_setup_roots_iterator(xcb_get_setup(xcb_connection)).rem;

    // int image_w = screen.width_in_pixels * screen_count;
    // int image_h = screen.height_in_pixels;

    // Obtain (properties of) gpu that does rendering.
    cuda_check_error(hipSetDevice(sr.gpu_id));

    // Setup OpenGL
    if(!glXMakeCurrent(dpy, sr.glx_window, sr.gl_context))
    {
        printf("Error: couldn't make context current for screen %d.\n",
               p.screen_no);
        exit(EXIT_FAILURE);
    }

    /*

        // Possibly bind OpenGL objects here

    */

    // Start compute+render loop
    int frame_no = 0;

    // To measure FPS
    benchmark_clock_t clock;
    benchmark_clock_t::time_point prev_swap_time = clock.now();
    while(program_should_run)
    {
        // Map OpenGL stuff into CUDA memory.
        if(p.use_interop)
        {
            /*
                cuda_check_error(hipGraphicsMapResources(..., ..., ...));
                size_t num_bytes;
                cuda_check_error(hipGraphicsResourceGetMappedPointer(..., &num_bytes, ...));
            */
        }

        // Compute


        // Render

        if(p.use_interop)
        {
            // We are done, just unmap OpenGL resources
            // cuda_check_error(hipGraphicsUnmapResources(..., ..., ...));

        }

        // glDraw...()

        // Ensure all outstanding OpenGL commands have been processed.
        glFinish();

        // Swap front and back buffer of the GLXWindow, first screen waits till
        // others wait to update view-state.
        int num_swappable = 0;
        if(p.screen_no == 0)
        {
            while(num_swappable != screen_count-1)
            {
                num_swappable = 0;
                for(int i = 0; i < screen_count; i++)
                    num_swappable += swap_ready.at(i);
            }
            vpar_render = vpar_event;
        }

        swap_ready.at(p.screen_no) = 1; // For view state only

        pthread_barrier_wait(&swap_barrier);
        glXSwapBuffers(dpy, sr.glx_window);

        swap_ready.at(p.screen_no) = 0;  // For view state only

        // Record some performance measurements
        benchmark_clock_t::duration d = clock.now() - prev_swap_time;
        inter_frame_durations.at(p.screen_no) = d;
        if(p.record_fps)
            inter_frame_durations_benchmark.at(p.screen_no).push_back(d);
        prev_swap_time = clock.now();

        frame_no += 1;
        if(frame_no == p.max_iterations)
            program_should_run = false;
    }

    release_gl_context(dpy);

    return NULL;
}

void map_windows()
{
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        screen_resources_t &sr = display_screen_resources.at(screen_no);

        // When using NVIDIA TwinView, a resize is triggered when setting the
        // fullscreen hint/property of an X window, which prevents it from
        // spanning the entire X screen. To circumvent we set the WM to ignore
        // the window.
        ignore_wm_redirect(xcb_connection, sr.x_window);

        xcb_map_window(xcb_connection, sr.x_window);
    }

    // Since the WM now ignores the window, we need to set input focus
    // manually.
    xcb_set_input_focus(xcb_connection, XCB_INPUT_FOCUS_NONE,
                        display_screen_resources[0].x_window, XCB_CURRENT_TIME);
}

// Mouse state
double mouse_prevx = 0.0;
double mouse_prevy = 0.0;

typedef struct elparameters_
{
    bool print_fps;
    bool print_path;
} elparameters_t;

void *event_loop(void *args)
{
    elparameters_t p = *((elparameters_t *)args);

    // Main Event loop: (busy) wait for input -> process -> output -> etc.
    while(program_should_run)
    {
        xcb_generic_event_t *event = xcb_wait_for_event(xcb_connection);
        if(!event)
        {
            printf("Error: event i/o error.\n");
            exit(EXIT_FAILURE);
        }

        if(event->response_type == XCB_EXPOSE)
        {}

        else if (event->response_type == XCB_MOTION_NOTIFY)
        {
            xcb_motion_notify_event_t *motion_event = (xcb_motion_notify_event_t*)event;
            int screen_no = window_to_screen_no[motion_event->root];
            float xpos = motion_event->root_x + screen_order[screen_no] * 1920;
            float ypos = motion_event->root_y;

            mouse_prevx = xpos;
            mouse_prevy = ypos;
        }

        else if(event->response_type == XCB_BUTTON_PRESS)
        {
            xcb_button_press_event_t *button_press_event = (xcb_button_press_event_t*)event;
            int screen_no = window_to_screen_no[button_press_event->root];

            // Mouse 1 click
            if(button_press_event->detail == XCB_BUTTON_INDEX_1)
            {
                mouse_prevx = button_press_event->root_x + screen_order[screen_no] * 1920;
                mouse_prevy = button_press_event->root_y;
            }

            // Scrolling
            else if(button_press_event->detail == XCB_BUTTON_INDEX_4)
            {
            }

            else if(button_press_event->detail == XCB_BUTTON_INDEX_5)
            {
            }

        }

        else if(event->response_type == XCB_KEY_PRESS)
        {
            xcb_key_press_event_t *key_press_event = (xcb_key_press_event_t*)event;
            xcb_keysym_t keysym = xcb_get_keysym(xcb_connection, key_press_event->detail);
            if(keysym == XK_equal) {}

            else if(keysym == XK_minus) {}

            else if(keysym == XK_Right) {}

            else if(keysym == XK_Left) {}

            else if(keysym == XK_Up) {}

            else if(keysym == XK_Down) {}

            // Exit app.
            else if(keysym == XK_q or keysym == XK_Escape)
                program_should_run = false;

        }

        else if(event->response_type == XCB_CONFIGURE_NOTIFY) {}

        fflush(stdout);
    }

    return NULL;
}

void *stat_reporter_loop(void *args)
{
    while(program_should_run)
    {
        std::this_thread::sleep_for(std::chrono::milliseconds{350});

        printf("FPS: <");
        for(auto it = x_screens.begin(); it != x_screens.end(); )
        {
            const int screen_no = it->first;
            printf("%.2f",
             1000.0 / to_ms(inter_frame_durations.at(screen_no)));
            if((++it) != x_screens.end()) printf(", ");
        }
        printf(">");
        fflush(stdout);
        printf("\r");
    }
    return NULL;
}

std::vector<int> get_render_gpu_ids()
{
    std::vector<int> render_gpu_ids;
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        const int screen_no = it.first;
        screen_resources_t &sr = display_screen_resources[screen_no];
        render_gpu_ids.push_back(sr.gpu_id);
    }
    return render_gpu_ids;
}

void run_gui(crlparameters_t crl_parameters, elparameters_t el_parameters,
             bool print_fps, bool benchmark)
{
    // Initiate a compute_render_loop thread for each screen
    pthread_barrier_init(&swap_barrier, NULL, x_screens.size());
    std::map<int, crlparameters_t> screen_parameters;
    pthread_t t;
    for(std::pair<int, xcb_screen_t> it : x_screens)
    {
        crlparameters_t p = crl_parameters;
        p.screen_no = it.first;
        screen_parameters[it.first] = p;
        pthread_create(&t, NULL, compute_render_loop, (void *) &screen_parameters[it.first]);
        threads.push_back(t);
    }

    if(print_fps)
    {
        pthread_create(&t, NULL, stat_reporter_loop, (void *) NULL);
        threads.push_back(t);
    }

    if(not benchmark)
    {
        pthread_create(&t, NULL, event_loop, (void *) &el_parameters);
        threads.push_back(t);
    }
    // Wait until threads are done.
    for(auto &t : threads) pthread_join(t, NULL);
}

int main(int argc, char const *argv[])
{
    bool benchmark = false;
    int benchmark_avg_factor = 25;
    bool use_interop = true;
    bool print_fps = false;
    for (int arg_no = 1; arg_no < argc; arg_no++)
    {
        if(std::string(argv[arg_no]) == "no_interop")
            use_interop = false;

        else if(std::string(argv[arg_no]) == "print_fps")
            print_fps = true;

        else if(std::string(argv[arg_no]) == "avg_factor")
        {
            benchmark_avg_factor = std::stoi(argv[arg_no+1]);
            arg_no += 1;
        }

        else
        {
            printf("Error: %s is not recognized as setting.\n", argv[arg_no]);
            exit(EXIT_FAILURE);
        }
    }

    init_xlib();
    init_xcb();
    load_glx(dpy);
    init_screens();
    create_xwindows();
    create_glxwindows();
    init_gl();
    map_windows();

    compute_gpu_ids = get_render_gpu_ids();

    if(use_interop) init_interop();

    // compute_render_loop
    crlparameters_t crl_commonp;
    crl_commonp.use_interop = use_interop;

    // event_loop
    elparameters_t el_commonp;
    el_commonp.print_fps = print_fps;

    crlparameters_t crl_parameters = crl_commonp;

    if(benchmark)
    {
        crl_parameters.record_fps = true;
        crl_parameters.max_iterations = benchmark_avg_factor;
        run_gui(crl_parameters, el_commonp, print_fps, true);

        // print inter_frame_durations, skip dummy (i.e. i = 0)
        for(int i = 1; i <= benchmark_avg_factor; ++i)
        {
            for(auto it = x_screens.begin(); it != x_screens.end(); ++it)
            {
                int screen_no = it->first;
                printf("%s%f", it != x_screens.begin() ? "," : "",
                               to_ms(inter_frame_durations_benchmark[screen_no][i]));
            }
            printf("\n");
        }
    }

    else
    {
        crl_parameters.record_fps = false;
        crl_parameters.max_iterations = 0;
        run_gui(crl_parameters, el_commonp, print_fps, false);
    }


    exit(EXIT_SUCCESS);
}
